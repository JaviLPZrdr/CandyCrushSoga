#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <cstdio>
#include <iostream>

using namespace std;

__global__ void convolucion2D(int* d_matriz,int fil,int col) {
	for (int i = 0; i < fil; i++) {
		for (int j = 0; j < col; j++) {
			printf("%d", d_matriz[i * col + j]);
		}
		printf("\n");
	}
}

void crearMatriz(int* matriz, int fil, int col, int dif) {
	if (dif == 1) {
		for (int j = 0; j < fil; j++) {

			for (int i = 0; i < col; i++) {

				matriz[i * col + j] = rand() % 4 + 1;
			}
		}
	}
	else {
		for (int j = 0; j < fil; j++) {

			for (int i = 0; i < col; i++) {

				matriz[i * col + j] = rand() % 6 + 1;
			}
		}
	}
}

void eliminarPosicion(int X, int Y, int** matriz) {
	matriz[X][Y] = 0;
}



int main() {
	int dif;
	int fil;
	int col;
	cout << "Dificultad (1 Facil y 2 Dificil)" << endl;
	cin >> dif;
	cout << "Tamanio de la matriz" << endl;
	cout << "Numero de columnas" << endl;
	cin >> col;
	cout << "Numero de filas" << endl;
	cin >> fil;
	printf("\n");
	int* matriz = new int[fil * col];

	crearMatriz(matriz, fil, col, dif);

	int* d_matriz;
	hipMalloc((void**)&d_matriz, fil * col * sizeof(int));



	dim3 numBloques(1);
	dim3 hilosEnBloque(1);

	hipMemcpy(d_matriz, matriz, col * fil * sizeof(int), hipMemcpyHostToDevice);
	
	convolucion2D << <numBloques, hilosEnBloque >> > (d_matriz, fil, col);
	
	hipMemcpy(matriz, d_matriz, col * fil * sizeof(int), hipMemcpyDeviceToHost);

	hipFree(d_matriz);
}
	/*
	* 
	* ////////////Codigo de 2D por si acaso
	hipMemcpy(d_matriz, matriz, col * sizeof(int), hipMemcpyHostToDevice);
	dim3 numBloques(1);
	dim3 hilosEnBloque(1);
	convolucion2D << <numBloques, hilosEnBloque >> > (d_matriz, fil, col);

	for (int i = 0; i < fil; i++) {
		hipMemcpy(matriz[i], d_matriz[i], col * sizeof(int), hipMemcpyDeviceToHost);
	}
	for (int i = 0; i < fil; i++) {
		hipFree(d_matriz[i]); // libera memoria para cada fila de la matriz
	}
	hipFree(d_matriz);
		/*for (int i = 0; i < fil; i++) {
		hipMalloc((void**)&d_matriz[i], col * sizeof(int));
	}*/

	/*for (int i = 0; i < fil; i++) {

		hipMemcpy(d_matriz[i], matriz[i], col * sizeof(int), hipMemcpyHostToDevice);
	}*/

		/*
		* 
		* 
		* 
		* /////Codigo lo de pillar las coordenadas
	int X_coord;
	int Y_coord;
	cout << "Fila en la que quieres borrar" << endl;
	cin >> X_coord;
	cout << "Columna en la que quieres borrar" << endl;
	cin >> Y_coord;
	eliminarPosicion(X_coord, Y_coord, matriz);
	for (int i = 0; i < fil; i++) {
		for (int j = 0; j < col; j++) {
			cout << matriz[i][j] << " ";
		}
		cout << endl;
	}
	*/

